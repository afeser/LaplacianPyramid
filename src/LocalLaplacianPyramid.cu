#include "hip/hip_runtime.h"
#include "ppm.hpp"
#include "pyramid.hpp"

__global__ void _llf(pixelByte *I, pixelByte *O, unsigned width, unsigned height, int fact, float ref, float sigma){
  int x = blockIdx.y*BLOCK_SIZE*width + blockIdx.x*BLOCK_SIZE + threadIdx.y*width + threadIdx.x; //current pixel

  O[x] = fact*(I[x] - ref) * expf(
                                - (I[x]-ref) * (I[x]-ref) / (2 * sigma * sigma)
                              );

}
__global__ void updateOutputLaplacian(pixelByte *tempLaplace, pixelByte *outLaplace, pixelByte *gaussian, unsigned width, unsigned height, float ref, float discretisation_step){
  int x = blockIdx.y*BLOCK_SIZE*width + blockIdx.x*BLOCK_SIZE + threadIdx.y*width + threadIdx.x; //current pixel

  outLaplace[x] += (fabsf(gaussian[x] - ref) < discretisation_step) *
                   tempLaplace[x] *
                   (1 - fabsf(gaussian[x] - ref) / discretisation_step);

}

__global__ void _upSample2      (pixelByte *in, pixelByte *out, int width, int height){
  int x = blockIdx.y*BLOCK_SIZE*width + blockIdx.x*BLOCK_SIZE + threadIdx.y*width + threadIdx.x; //current pixel


  if(x > width*height-1){
    return;
  }

  out[(x%width)*2 + x/width*(width*4)] = in[x];

  if((x+1)%width != 0){ // 1 sagi
    // No row change
    out[(x%width)*2 + x/width*(width*4)+1] = (in[x+1] + in[x]) / 2;
  }else{
    out[(x%width)*2 + x/width*(width*4)+1] = in[x];
  }
  if((x+width)/width < height){ // 1 alti
    out[(x%width)*2 + x/width*(width*4)+width*2] = (in[x+width] + in[x]) / 2;
  }else{
    out[(x%width)*2 + x/width*(width*4)+width*2] = in[x];
  }
  if((x+width)/width < height && (x+1)%width != 0){
    out[(x%width)*2 + x/width*(width*4)+1+width*2] = (in[x+1] + in[x] + in[x+width] + in[x+width+1]) / 4; // 1 sagi 1 alti
  }else{
    out[(x%width)*2 + x/width*(width*4)+1+width*2] = in[x];
  }
}
__global__ void _setLaplacian   (pixelByte *inPic, pixelByte *laplacian, unsigned width, unsigned height){
  /*
   * Set laplacian to recover the deleted data for Gaussian filter, not reduced size.
   */

  int x = blockIdx.y*BLOCK_SIZE*width + blockIdx.x*BLOCK_SIZE + threadIdx.y*width + threadIdx.x; //current pixel

  inPic[x] = inPic[x] - laplacian[x];


}

void localLaplacianPyramid(char *inputPath,
                           char *outputPath,
                           const float sigma,
                           const int pyramidHeight,
                           const int fact,
                           const int N
                           ){


  Picture inPic(inputPath, true);

  const float discretisation_step =  1.0 /  (N-1); // linespace tanimi boyle cunku

  Pyramid *gaussianP = new Pyramid();
  Pyramid *outputP = new Pyramid();

  gaussianP->createGaussian(&inPic, pyramidHeight);

  outputP->createLaplacian(&inPic, pyramidHeight);

  for(float ref = 0; ref<=1; ref+=discretisation_step){
    // Map to a new image
    dim3 dimBlock2(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid2 (inPic.width/BLOCK_SIZE, inPic.height/BLOCK_SIZE);
    Picture mapped(inPic.width, inPic.height, true);

    // Converting the base image to a new mapped image
    _llf<<<dimGrid2, dimBlock2>>>(inPic.R, mapped.R, inPic.width, inPic.height, fact, ref, sigma);
    _llf<<<dimGrid2, dimBlock2>>>(inPic.G, mapped.G, inPic.width, inPic.height, fact, ref, sigma);
    _llf<<<dimGrid2, dimBlock2>>>(inPic.B, mapped.B, inPic.width, inPic.height, fact, ref, sigma);

    // Find new Laplacian Pyramid from the mapped image
    Pyramid tempLaplacian;
    tempLaplacian.createLaplacian(&mapped, pyramidHeight);

    // Do for all layers
    for(int l = 0; l<pyramidHeight; l++){
      unsigned width  = inPic.width /std::pow(2, l);
      unsigned height = inPic.height/std::pow(2, l);

      dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
      dim3 dimGrid (width/BLOCK_SIZE, height/BLOCK_SIZE);

      updateOutputLaplacian<<<dimGrid, dimBlock>>>(tempLaplacian.getLayer(l)->R, outputP->getLayer(l)->R, gaussianP->getLayer(l)->R, width, height, ref, discretisation_step);
      updateOutputLaplacian<<<dimGrid, dimBlock>>>(tempLaplacian.getLayer(l)->G, outputP->getLayer(l)->G, gaussianP->getLayer(l)->G, width, height, ref, discretisation_step);
      updateOutputLaplacian<<<dimGrid, dimBlock>>>(tempLaplacian.getLayer(l)->B, outputP->getLayer(l)->B, gaussianP->getLayer(l)->B, width, height, ref, discretisation_step);
    }

  }
  Pyramid piramit = Pyramid();
  piramit.createLaplacian(&inPic, 5);
  piramit.getLayer(0)->write("GERCEKLAPLAS.ppm");
  outputP->getLayer(0)->write("LAPLACIAN.ppm");

  // Collapse the pyramid
  for(int i = pyramidHeight-1; i > 0; i--){
    unsigned width  = gaussianP->getLayer(i-1)->width;
    unsigned height = gaussianP->getLayer(i-1)->height;

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid (width/2/BLOCK_SIZE, height/2/BLOCK_SIZE);

    dim3 dimBlock2(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid2 (width/BLOCK_SIZE, height/BLOCK_SIZE);

    _upSample2<<<dimGrid, dimBlock>>>(gaussianP->getLayer(i)->R, gaussianP->getLayer(i-1)->R, width/2, height/2);
    _upSample2<<<dimGrid, dimBlock>>>(gaussianP->getLayer(i)->G, gaussianP->getLayer(i-1)->G, width/2, height/2);
    _upSample2<<<dimGrid, dimBlock>>>(gaussianP->getLayer(i)->B, gaussianP->getLayer(i-1)->B, width/2, height/2);

    _setLaplacian<<<dimGrid2, dimBlock2>>>(gaussianP->getLayer(i-1)->R, outputP->getLayer(i-1)->R, width, height);
    _setLaplacian<<<dimGrid2, dimBlock2>>>(gaussianP->getLayer(i-1)->G, outputP->getLayer(i-1)->G, width, height);
    _setLaplacian<<<dimGrid2, dimBlock2>>>(gaussianP->getLayer(i-1)->B, outputP->getLayer(i-1)->B, width, height);
  }
  gaussianP->getLayer(0)->write(outputPath);

}
