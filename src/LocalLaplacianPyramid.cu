#include "hip/hip_runtime.h"
#include "ppm.hpp"
#include "pyramid.hpp"

__global__ void _r           (pixelByte *I, pixelByte *O, pixelByte g, float sigma, float alpha, unsigned width, unsigned height){
  // f function is taken polinomial(at least a power function)
  int x = blockIdx.y*BLOCK_SIZE*width + blockIdx.x*BLOCK_SIZE + threadIdx.y*width + threadIdx.x; //current pixel

  int i;
  if(I[x] < g){
    i = g - sigma*powf( (g - I[x]) / sigma, alpha);
  }else{
    i = g + sigma*powf( (I[x] - g) / sigma, alpha);
  }
  if(i > 255){
    O[x] = 255;
  }else if(i < 0){
    O[x] = 0;
  }else{
    O[x] = i;
  }
}
__global__ void _upSample2    (pixelByte *in, pixelByte *out, int width, int height){
  int x = blockIdx.y*BLOCK_SIZE*width + blockIdx.x*BLOCK_SIZE + threadIdx.y*width + threadIdx.x; //current pixel


  if(x > width*height-1){
    return;
  }

  out[(x%width)*2 + x/width*(width*4)] = in[x];

  if((x+1)%width != 0){ // 1 sagi
    // No row change
    out[(x%width)*2 + x/width*(width*4)+1] = (in[x+1] + in[x]) / 2;
  }else{
    out[(x%width)*2 + x/width*(width*4)+1] = in[x];
  }
  if((x+width)/width < height){ // 1 alti
    out[(x%width)*2 + x/width*(width*4)+width*2] = (in[x+width] + in[x]) / 2;
  }else{
    out[(x%width)*2 + x/width*(width*4)+width*2] = in[x];
  }
  if((x+width)/width < height && (x+1)%width != 0){
    out[(x%width)*2 + x/width*(width*4)+1+width*2] = (in[x+1] + in[x] + in[x+width] + in[x+width+1]) / 4; // 1 sagi 1 alti
  }else{
    out[(x%width)*2 + x/width*(width*4)+1+width*2] = in[x];
  }
}
__global__ void _setLaplacian(pixelByte *inPic, pixelByte *laplacian, unsigned width, unsigned height){
  /*
   * Set laplacian to recover the deleted data for Gaussian filter, not reduced size.
   */

  int x = blockIdx.y*BLOCK_SIZE*width + blockIdx.x*BLOCK_SIZE + threadIdx.y*width + threadIdx.x; //current pixel

  int i;
  i = inPic[x] + laplacian[x] - 128;
  if(i < 0){
   inPic[x] = 0;
  }else if(i > 255){
   inPic[x] = 255;
  }else{
   inPic[x] = i;
  }

}

void localLaplacianPyramid(char *inputPath,
                           char *outputPath,
                           const float sigma,
                           const float alpha,
                           const int pyramidHeight,
                           const int number_of_additions){


  Picture inPic(inputPath, true);


  Pyramid *gaussianP = new Pyramid();
  Pyramid *laplacianP = new Pyramid();
  Pyramid *outputP = new Pyramid();

  gaussianP->createGaussian(&inPic, pyramidHeight); // COOOK GARIIP!!!! biz buna veri yolluyoruz ama yolladigimiz objenin destructor fonksiyonu bu fonksiyon bittiginde de cagiriliyor!!!!
  laplacianP->createLaplacian(&inPic, pyramidHeight);

  outputP->createLaplacian(&inPic, pyramidHeight);

  for(int l = 0; l<pyramidHeight; l++){

    unsigned width  = laplacianP->getLayer(l)->width;
    unsigned height = laplacianP->getLayer(l)->height;

    for(int y = 0; y<height; y++){
      for(int x = 0; x<width; x++){
        // Get Gaussian average for each layer
        Pixel g = gaussianP->getLayer(l)->getPixel(x, y);

        // Map to a new image
        dim3 dimBlock2(BLOCK_SIZE, BLOCK_SIZE);
        dim3 dimGrid2 (inPic.width/BLOCK_SIZE, inPic.height/BLOCK_SIZE);
        Picture mapped(inPic.width, inPic.height, true);

        // Converting the base image to a new mapped image
        _r<<<dimGrid2, dimBlock2>>>(inPic.R, mapped.R, g.R, sigma, alpha, inPic.width, inPic.height);
        _r<<<dimGrid2, dimBlock2>>>(inPic.G, mapped.G, g.G, sigma, alpha, inPic.width, inPic.height);
        _r<<<dimGrid2, dimBlock2>>>(inPic.B, mapped.B, g.B, sigma, alpha, inPic.width, inPic.height);

        // Find new Laplacian Pyramid for the mapped image
        Pyramid nLaplacianP;
        nLaplacianP.createLaplacian(&mapped, l+1); // burasi cooook buyuk memory kaplayacak is bittikten sonra silmezsek!!!

        // Update output pyramid
        Pixel p = nLaplacianP.getLayer(l)->getPixel(x, y);
        outputP->getLayer(l)->setPixel(x, y, p);
      }
    }
  }


  // Collapse the pyramid
  for(int i = pyramidHeight-1; i > 0; i--){
    unsigned width  = gaussianP->getLayer(i-1)->width;
    unsigned height = gaussianP->getLayer(i-1)->height;

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid (width/2/BLOCK_SIZE, height/2/BLOCK_SIZE);

    dim3 dimBlock2(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid2 (width/BLOCK_SIZE, height/BLOCK_SIZE);

    _upSample2<<<dimGrid, dimBlock>>>(gaussianP->getLayer(i)->R, gaussianP->getLayer(i-1)->R, width/2, height/2);
    _upSample2<<<dimGrid, dimBlock>>>(gaussianP->getLayer(i)->G, gaussianP->getLayer(i-1)->G, width/2, height/2);
    _upSample2<<<dimGrid, dimBlock>>>(gaussianP->getLayer(i)->B, gaussianP->getLayer(i-1)->B, width/2, height/2);

    for(int z = 0; z<number_of_additions; z++){
      _setLaplacian<<<dimGrid2, dimBlock2>>>(gaussianP->getLayer(i-1)->R, outputP->getLayer(i-1)->R, width, height);
      _setLaplacian<<<dimGrid2, dimBlock2>>>(gaussianP->getLayer(i-1)->G, outputP->getLayer(i-1)->G, width, height);
      _setLaplacian<<<dimGrid2, dimBlock2>>>(gaussianP->getLayer(i-1)->B, outputP->getLayer(i-1)->B, width, height);
    }
  }
  gaussianP->getLayer(0)->write(outputPath);

}
