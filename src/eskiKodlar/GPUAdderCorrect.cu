#include<stdio.h>
#include<stdlib.h>


// CUDA runtime
#include <hip/hip_runtime.h>



__global__ void add(int *result, int *num1, int *num2){
	*result = *num1 + *num2;
}


int main(int argc, char *argv[]){
	if(argc < 3){
		return 1;
	}

	int num1, num2, result;
	int *num1G, *num2G, *resultG;

	hipMalloc((void **)&num1G, sizeof(int));
	hipMalloc((void **)&num2G, sizeof(int));
	hipMalloc((void **)&resultG, sizeof(int));

	num1 = atoi(argv[1]);
	num2 = atoi(argv[2]);

	hipMemcpy(num1G, &num1, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(num2G, &num2, sizeof(int), hipMemcpyHostToDevice);

	add<<<1,1>>>(resultG, num1G, num2G);

	hipMemcpy(&result, resultG, sizeof(int), hipMemcpyDeviceToHost); // ilk parametre hep ustune yazilacak olan

	hipFree(num1G);
	hipFree(num2G);
	hipFree(resultG);


	printf("Result is : %d\n", result);

	return 0;
}
